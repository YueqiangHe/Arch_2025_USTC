// #include <__clang_cuda_builtin_vars.h>
// #include <__clang_cuda_runtime_wrapper.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_vector_types.h>
#define smalloc(type, ptr, num)                                                \
  if (!(ptr = (type *)malloc(sizeof(type) * (num))))                           \
  exit(1)
#define Blocksize (32)
#define Matsize (4096)
#define Verifysize (1024)
#define T (128)
#define U (16)
#define S (T / U)
__global__ void Matmul1(float *A, float *B, float *C, unsigned N) {
  unsigned row = blockIdx.y * blockDim.y + threadIdx.y;
  unsigned col = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned k;
  float sum = 0;
  for (k = 0; k < N; k++) {
    sum += A[row * N + k] * B[k * N + col];
  }
  C[row * N + col] = sum;
}

__global__ void Matmul2(float *A, float *B, float *C,
                        unsigned N) { // A,B with padding
  unsigned tx = threadIdx.x, ty = threadIdx.y;
  unsigned bx = blockIdx.x, by = blockIdx.y;
  unsigned row = by * blockDim.y + ty;
  unsigned col = bx * blockDim.x + tx;
  __shared__ float Asub[Blocksize][Blocksize], Bsub[Blocksize][Blocksize];
  float sum = 0;
  unsigned kk, k;
  for (kk = 0; kk < N; kk += Blocksize) {
    Asub[ty][tx] = A[row * N + (kk + tx)];
    Bsub[ty][tx] = B[(kk + ty) * N + col];
    __syncthreads();
    for (k = 0; k < Blocksize; k++) {
      sum += Asub[ty][k] * Bsub[k][tx];
    }
    __syncthreads();
  }
  C[row * N + col] = sum;
}

__global__ void Matmul3(float *A, float *B, float *C, unsigned N) {
  unsigned tid = threadIdx.x;
  unsigned c_row = blockIdx.y * T + tid;
  unsigned c_col = blockIdx.x * U;

  float Areg[S];
  float Creg[U] = {0};

  __shared__ float Bsub[S][U];

  for (unsigned kk = 0; kk < N; kk += S) {
// 把A的S个元素加载到寄存器中
#pragma unroll
    for (unsigned i = 0; i < S; i++) {
      Areg[i] = A[c_row * N + (kk + i)];
    }

    // 把B的S行U列加载到shared memory中
    unsigned b_row = tid / U;
    unsigned b_col = tid % U;
    Bsub[b_row][b_col] = B[(kk + b_row) * N + (c_col + b_col)];

    __syncthreads();

// 计算C的U个元素
#pragma unroll
    for (unsigned i = 0; i < S; i++) {
      float a_val = Areg[i];
#pragma unroll
      for (unsigned j = 0; j < U; j++) {
        Creg[j] += a_val * Bsub[i][j];
      }
    }

    __syncthreads();
  }

#pragma unroll
  for (unsigned j = 0; j < U; j++) {
    C[c_row * N + (c_col + j)] = Creg[j];
  }
}

__host__ void matmubase(float *A, float *B, float *C, unsigned N) {
  unsigned i, j, k;
  for (i = 0; i < N; i++) {
    for (j = 0; j < N; j++) {
      C[i * N + j] = 0;
      for (k = 0; k < N; k++) {
        C[i * N + j] += A[i * N + k] * B[k * N + j];
      }
    }
  }
}

__host__ void gen_mat(float **pA, float **pB, unsigned N) {
  float *A, *B;
  smalloc(float, A, N *N);
  smalloc(float, B, N *N);
  unsigned i;
  for (i = 0; i < N * N; i++) {
    A[i] = 1.0 * rand() / RAND_MAX;
    B[i] = 1.0 * rand() / RAND_MAX;
  }
  *pA = A;
  *pB = B;
}

__host__ unsigned compare(float *pred_, float *true_, unsigned n) {
  unsigned i;
  float relative_error;
  for (i = 0; i < n; i++) {
    relative_error = fabs((pred_[i] - true_[i]) / true_[i]);
    if (relative_error >= 1e-6) {
      printf("not equal! relative error: %12.9lf pred: %12.9f true: %12.9f\n",
             relative_error, pred_[i], true_[i]);
      return 1;
    }
  }
  printf("equal!\n");
  return 0;
}

int main(void) {
  const unsigned PN = Matsize, VN = Verifysize;
  float *hA, *hB, *hC1, *hC2, *dA, *dB, *dC1, *dC2, *Cbase;
  float *hC3, *dC3;
  gen_mat(&hA, &hB, VN);
  smalloc(float, Cbase, sizeof(float) * VN * VN);
  smalloc(float, hC1, sizeof(float) * VN * VN);
  smalloc(float, hC2, sizeof(float) * VN * VN);
  smalloc(float, hC3, sizeof(float) * VN * VN);
  hipMalloc(&dA, sizeof(float) * VN * VN);
  hipMalloc(&dB, sizeof(float) * VN * VN);
  hipMalloc(&dC1, sizeof(float) * VN * VN);
  hipMalloc(&dC2, sizeof(float) * VN * VN);
  hipMalloc(&dC3, sizeof(float) * VN * VN);
  hipMemcpy(dA, hA, sizeof(float) * VN * VN, hipMemcpyHostToDevice);
  hipMemcpy(dB, hB, sizeof(float) * VN * VN, hipMemcpyHostToDevice);

  dim3 gridsize(VN / Blocksize, VN / Blocksize),
      blocksize(Blocksize, Blocksize);
  Matmul1<<<gridsize, blocksize>>>(dA, dB, dC1, VN);
  Matmul2<<<gridsize, blocksize>>>(dA, dB, dC2, VN);
  Matmul3<<<dim3(VN / U, VN / T), T>>>(dA, dB, dC3, VN);
  hipMemcpy(hC1, dC1, sizeof(float) * VN * VN, hipMemcpyDeviceToHost);
  hipMemcpy(hC2, dC2, sizeof(float) * VN * VN, hipMemcpyDeviceToHost);
  hipMemcpy(hC3, dC3, sizeof(float) * VN * VN, hipMemcpyDeviceToHost);
  matmubase(hA, hB, Cbase, VN);
  hipDeviceSynchronize();

  int flag = 0;
  flag |= compare(hC1, Cbase, VN * VN);
  flag |= compare(hC2, Cbase, VN * VN);
  flag |= compare(hC3, Cbase, VN * VN);
  if (flag) {
    printf("error!\n");
    exit(1);
  }
  printf("pass!\n");
  free(hA);
  free(hB);
  free(hC1);
  free(hC2);
  free(Cbase);
  hipFree(dA);
  hipFree(dB);
  hipFree(dC1);
  hipFree(dC2);

  gen_mat(&hA, &hB, PN);
  hipMalloc(&dA, sizeof(float) * PN * PN);
  hipMalloc(&dB, sizeof(float) * PN * PN);
  hipMalloc(&dC1, sizeof(float) * PN * PN);
  hipMalloc(&dC2, sizeof(float) * PN * PN);
  hipMemcpy(dA, hA, sizeof(float) * PN * PN, hipMemcpyHostToDevice);
  hipMemcpy(dB, hB, sizeof(float) * PN * PN, hipMemcpyHostToDevice);

  gridsize = {PN / Blocksize, PN / Blocksize};
  blocksize = {Blocksize, Blocksize};

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float Time1 = 0.0, Time2 = 0.0, Time3 = 0.0, temp = 0;
  const unsigned loopnum = 10;
  unsigned i;
  for (i = 0; i < loopnum; i++) {

    hipEventRecord(start, 0);
    Matmul1<<<gridsize, blocksize>>>(dA, dB, dC1, PN);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&temp, start, stop);
    Time1 += temp;
    temp = 0;

    hipDeviceSynchronize();

    hipEventRecord(start, 0);
    Matmul2<<<gridsize, blocksize>>>(dA, dB, dC2, PN);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&temp, start, stop);
    Time2 += temp;
    temp = 0;

    hipDeviceSynchronize();

    hipEventRecord(start, 0);
    Matmul3<<<dim3(PN / U, PN / T), T>>>(dA, dB, dC3, PN);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&temp, start, stop);
    Time3 += temp;
    temp = 0;

    hipDeviceSynchronize();
  }

  printf("N: %5.d  time1: %12.9f  time2: %12.9f time3: %12.9f\n", PN,
         Time1 / loopnum, Time2 / loopnum, Time3 / loopnum);
  free(hA);
  free(hB);
  hipFree(dA);
  hipFree(dB);
  hipFree(dC1);
  hipFree(dC2);
}